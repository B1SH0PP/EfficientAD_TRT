#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "preprocess.h"

static uint8_t* img_buffer_host = nullptr;
static uint8_t* img_buffer_device = nullptr;

struct AffineMatrix {
  float value[6];
};

void preprocessImg(cv::Mat& img, int newh, int neww) {
  // bgr to rgb
  cv::cvtColor(img, img, cv::COLOR_BGR2RGB);
  // resize into 256
  cv::resize(img, img, cv::Size(neww, newh));
  //
  img.convertTo(img, CV_32FC3);
  // ImageNet normalize
  img /= 255.0f;
  img -= cv::Scalar(0.485, 0.456, 0.406);
  img /= cv::Scalar(0.229, 0.224, 0.225);
}

__global__ void warpaffine_kernel(uint8_t* src, int src_line_size, int src_width, int src_height,
                                  float* dst, int dst_width, int dst_height, uint8_t const_value_st,
                                  AffineMatrix d2s, int edge) {
  /*
  src: img_buffer_device
  src_line_size: src_width * 3
  dst: gpu_buffers[0]
  const_value_st: 128
  edge: jobs 256
  */
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= edge) return;

  float m_x1 = d2s.value[0];
  float m_y1 = d2s.value[1];
  float m_z1 = d2s.value[2];
  float m_x2 = d2s.value[3];
  float m_y2 = d2s.value[4];
  float m_z2 = d2s.value[5];

  // Calculate the position of each pixel on the output image corresponding to the input image based on
  // the affine transformation matrix
  int dx = position % dst_width;
  int dy = position / dst_width;
  float src_x = m_x1 * dx + m_y1 * dy + m_z1 + 0.5f;
  float src_y = m_x2 * dx + m_y2 * dy + m_z2 + 0.5f;
  float c0, c1, c2;

  // If the input image position out of range, fill the output image with a constant
  if (src_x <= -1 || src_x >= src_width || src_y <= -1 || src_y >= src_height) {
    // out of range
    c0 = const_value_st;
    c1 = const_value_st;
    c2 = const_value_st;
  } else {
    // bilinear interpolation calculates the output image pixel
    int y_low = floorf(src_y);
    int x_low = floorf(src_x);
    int y_high = y_low + 1;
    int x_high = x_low + 1;

    uint8_t const_value[] = {const_value_st, const_value_st, const_value_st};
    float ly = src_y - y_low;
    float lx = src_x - x_low;
    float hy = 1 - ly;
    float hx = 1 - lx;
    float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
    uint8_t* v1 = const_value;
    uint8_t* v2 = const_value;
    uint8_t* v3 = const_value;
    uint8_t* v4 = const_value;

    if (y_low >= 0) {
      if (x_low >= 0)
        v1 = src + y_low * src_line_size + x_low * 3;

      if (x_high < src_width)
        v2 = src + y_low * src_line_size + x_high * 3;
    }

    if (y_high < src_height) {
      if (x_low >= 0)
        v3 = src + y_high * src_line_size + x_low * 3;

      if (x_high < src_width)
        v4 = src + y_high * src_line_size + x_high * 3;
    }

    // `w1/w2/w3/w4` means the weight of bilinear interpolation
    // `v1/v2/v3/v4` means the adjacent pixel value(4 pixels) of the input image
    // 4 pixels -> 1 pixel
    c0 = w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0];  // B
    c1 = w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1];  // G
    c2 = w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2];  // R
  }

  // bgr to rgb
  float t = c2;
  c2 = c0;
  c0 = t;

  // normalization [0,1]
  c0 = c0 / 255.0f;
  c1 = c1 / 255.0f;
  c2 = c2 / 255.0f;

  // rgbrgbrgb to rrrgggbbb
  int area = dst_width * dst_height;
  float* pdst_c0 = dst + dy * dst_width + dx;
  float* pdst_c1 = pdst_c0 + area;
  float* pdst_c2 = pdst_c1 + area;
  *pdst_c0 = c0;
  *pdst_c1 = c1;
  *pdst_c2 = c2;
}

void cuda_preprocess(uint8_t* src, int src_width, int src_height, float* dst, int dst_width, int dst_height,
                     hipStream_t stream) {
  /*
  dst: gpu_buffers
   cv::Mat -> .ptr() -> uint8_t* src -> memcpy() -> uint8_t* img_buffer_host -> hipMemcpyAsync() ->
   uint8_t* img_buffer_device
  */
  int img_size = src_width * src_height * 3;
  // copy data to pinned memory
  memcpy(img_buffer_host, src, img_size);
  // copy data to device memory
  CUDA_CHECK(hipMemcpyAsync(img_buffer_device, img_buffer_host, img_size, hipMemcpyHostToDevice, stream));

  /* deprecated */
  AffineMatrix s2d, d2s;
  float scale = std::min(dst_height / (float)src_height, dst_width / (float)src_width);

  s2d.value[0] = scale;
  s2d.value[1] = 0;
  s2d.value[2] = -scale * src_width * 0.5 + dst_width * 0.5;
  s2d.value[3] = 0;
  s2d.value[4] = scale;
  s2d.value[5] = -scale * src_height * 0.5 + dst_height * 0.5;

  cv::Mat m2x3_s2d(2, 3, CV_32F, s2d.value);
  cv::Mat m2x3_d2s(2, 3, CV_32F, d2s.value);
  cv::invertAffineTransform(m2x3_s2d, m2x3_d2s);  // to get the invert affine matrix -> m2x3_d2s

  memcpy(d2s.value, m2x3_d2s.ptr<float>(0), sizeof(d2s.value));  // m2x3_d2s.ptr<float>(0):return a float pointer to the first row(`0`) of the Mat

  int jobs = dst_height * dst_width;
  int threads = 256;
  int blocks = ceil(jobs / (float)threads);

  warpaffine_kernel<<<blocks, threads, 0, stream>>>(img_buffer_device, src_width * 3, src_width, src_height,
                                                    dst, dst_width, dst_height, 128, d2s, jobs);
}

void cuda_batch_preprocess(std::vector<cv::Mat>& img_batch, float* dst, int dst_width, int dst_height,
                           hipStream_t stream) {
  /*
  float* dst = gpu_buffers[0]
  */
  int dst_size = dst_width * dst_height * 3;
  for (size_t i = 0; i < img_batch.size(); i++) {
    // dst[dst_size * i]: each image's first address in gpu_buffers
    cuda_preprocess(img_batch[i].ptr(), img_batch[i].cols, img_batch[i].rows, &dst[dst_size * i],
                    dst_width, dst_height, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));
  }
}

void cuda_preprocess_init(int max_image_size) {
  // prepare input data in pinned memory
  CUDA_CHECK(hipHostMalloc((void**)&img_buffer_host, max_image_size * 3));
  // prepare input data in device memory
  CUDA_CHECK(hipMalloc((void**)&img_buffer_device, max_image_size * 3));
}

void cuda_preprocess_destroy() {
  CUDA_CHECK(hipFree(img_buffer_device));
  CUDA_CHECK(hipHostFree(img_buffer_host));
}
